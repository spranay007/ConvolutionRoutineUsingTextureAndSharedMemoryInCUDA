#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "convolutionTexture_common.h"

// Define kernel parameters
#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 32

// Round a / b to nearest higher integer value
inline int iDivUp(int a, int b) { return (a % b != 0) ? (a / b + 1) : (a / b); }

// Constant memory for the convolution kernel
__constant__ float c_Kernel[KERNEL_LENGTH_MAX];

// Function to set the convolution kernel in constant memory
extern "C" void setConvolutionKernel(float* h_Kernel, int kernelLengthUser) {
    hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel), h_Kernel, kernelLengthUser * sizeof(float));
}

// Kernel for row convolution
__global__ void convolutionRowsKernel(float* d_Dst, int imageW, int imageH, hipTextureObject_t texSrc, int kernelRadiusUser) {
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;

    // Define shared memory array
    __shared__ float sharedMemory[BLOCK_SIZE_Y][BLOCK_SIZE_X + 2 * KERNEL_RADIUS_MAX];
    if (ix < imageW && iy < imageH) {
        // Compute global index
        int globalIdx = iy * imageW + ix;

        // Load data into shared memory
        sharedMemory[threadIdx.y][threadIdx.x + kernelRadiusUser] = tex2D<float>(texSrc, ix + 0.5f, iy + 0.5f);

        // Load ghost elements into shared memory
        if (threadIdx.x < kernelRadiusUser) {
            sharedMemory[threadIdx.y][threadIdx.x] = tex2D<float>(texSrc, ix - kernelRadiusUser + 0.5f, iy + 0.5f);
        }
        if (threadIdx.x >= blockDim.x - kernelRadiusUser) {
            sharedMemory[threadIdx.y][threadIdx.x + 2 * kernelRadiusUser] = tex2D<float>(texSrc, ix + blockDim.x - kernelRadiusUser + 0.5f, iy + 0.5f);
        }

        // Synchronize threads to ensure all data is loaded into shared memory
        __syncthreads();

        float sum = 0;

        // Perform convolution using data from shared memory
        for (int k = -kernelRadiusUser; k <= kernelRadiusUser; k++) {
            sum += sharedMemory[threadIdx.y][threadIdx.x + kernelRadiusUser + k] * c_Kernel[kernelRadiusUser - k];
        }

        d_Dst[globalIdx] = sum;
    }
}

// Wrapper function for row convolution GPU kernel
extern "C" void convolutionRowsGPU(float* d_Dst, hipArray * a_Src, int imageW, int imageH, hipTextureObject_t texSrc, int kernelRadiusUser) {
    dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 blocks(iDivUp(imageW, threads.x), iDivUp(imageH, threads.y));

    convolutionRowsKernel <<<blocks, threads >>> (d_Dst, imageW, imageH, texSrc, kernelRadiusUser);
    getLastCudaError("convolutionRowsKernel() execution failed\n");
}

// Kernel for column convolution
__global__ void convolutionColumnsKernel(float* d_Dst, int imageW, int imageH, hipTextureObject_t texSrc, int kernelRadiusUser) {
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;

    // Define shared memory array
    __shared__ float sharedMemory[BLOCK_SIZE_Y + 2 * KERNEL_RADIUS_MAX][BLOCK_SIZE_X];

    if (ix < imageW && iy < imageH) {
        // Compute global index
        int globalIdx = iy * imageW + ix;

        // Load data into shared memory
        sharedMemory[threadIdx.y + kernelRadiusUser][threadIdx.x] = tex2D<float>(texSrc, ix + 0.5f, iy + 0.5f);

        // Load ghost elements into shared memory
        if (threadIdx.y < kernelRadiusUser) {
            sharedMemory[threadIdx.y][threadIdx.x] = tex2D<float>(texSrc, ix + 0.5f, iy - kernelRadiusUser + 0.5f);
        }
        if (threadIdx.y >= blockDim.y - kernelRadiusUser) {
            sharedMemory[threadIdx.y + 2 * kernelRadiusUser][threadIdx.x] = tex2D<float>(texSrc, ix + 0.5f, iy + blockDim.y - kernelRadiusUser + 0.5f);
        }

        // Synchronize threads to ensure all data is loaded into shared memory
        __syncthreads();

        float sum = 0;

        // Perform convolution using data from shared memory
        for (int k = -kernelRadiusUser; k <= kernelRadiusUser; k++) {
            sum += sharedMemory[threadIdx.y + kernelRadiusUser + k][threadIdx.x] * c_Kernel[kernelRadiusUser - k];
        }

        d_Dst[globalIdx] = sum;
    }
}

// Wrapper function for column convolution GPU kernel
extern "C" void convolutionColumnsGPU(float* d_Dst, hipArray * a_Src, int imageW, int imageH, hipTextureObject_t texSrc, int kernelRadiusUser) {
    dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 blocks(iDivUp(imageW, threads.x), iDivUp(imageH, threads.y));

    convolutionColumnsKernel <<<blocks, threads >>> (d_Dst, imageW, imageH, texSrc, kernelRadiusUser);
    getLastCudaError("convolutionColumnsKernel() execution failed\n");
}
